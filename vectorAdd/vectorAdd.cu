#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<iostream>
__global__ void vectorAdd(const float *input_vecs, float *output_vec, size_t n, size_t dim){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if(i<n && j<dim) atomicAdd(&output_vec[j], input_vecs[i * dim + j]);
}
void checkCudaError(hipError_t err, const char* msg){
	if(err != hipSuccess){
		std::cerr << "CUDA Error: " << msg << "-" << hipGetErrorString(err) << std::endl;
		exit(EXIT_FAILURE);
	}
}
void reduce_sum(const float* input_vecs, size_t n, size_t dim, float* output_vec){
	float *cuda_input_vecs, *cuda_output_vec;
	checkCudaError(hipMalloc((void**)&cuda_input_vecs,n*dim*sizeof(float)), "hipMalloc cuda_input_vecs");
	checkCudaError(hipMalloc((void**)&cuda_output_vec,dim*sizeof(float)), "hipMalloc cuda_output_vec");
	for(int i=0;i<dim;++i) output_vec[i]=0;
	checkCudaError(hipMemcpy(cuda_input_vecs,input_vecs,n*dim*sizeof(float),hipMemcpyHostToDevice), "cudaCopy input_vecs to cuda_input_vecs");
	checkCudaError(hipMemcpy(cuda_output_vec,output_vec,dim*sizeof(float),hipMemcpyHostToDevice), "cudaCopy output_vec to cuda_output_vec");
	dim3 threadsPerBlock(16,16);
	dim3 numBlocks((n+threadsPerBlock.x-1) / threadsPerBlock.x, (dim+threadsPerBlock.y-1)/threadsPerBlock.y);
	vectorAdd<<<numBlocks, threadsPerBlock>>>(cuda_input_vecs, cuda_output_vec, n, dim);
	checkCudaError(hipMemcpy(output_vec,cuda_output_vec,dim*sizeof(float),hipMemcpyDeviceToHost), "cudaCopy cuda_output_vec to output_vec");
	hipFree(cuda_input_vecs);
	hipFree(cuda_output_vec);
}
int main(){
	int n=1000,dim=1000;
	float *a,*b;
	a=(float*)malloc(n*dim*sizeof(float));
	b=(float*)malloc(dim*sizeof(float));
	for(int i=0;i<n;++i){
		for(int j=0;j<dim;++j){
			a[i*dim+j]=i*j;
		}
	}
	reduce_sum(a,n,dim,b);
	for(int i=0;i<min(dim,10);++i) std::cout<<b[i]<<' ';
	std::cout<<'\n';
	return 0;
}